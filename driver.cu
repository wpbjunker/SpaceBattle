#include "hip/hip_runtime.h"
#include <math.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <SDL.h>
#include <hip/hip_runtime.h>

#include "driver.h"
#include "board.h"
/************************MACRO DEFINITIONS**********************************/


#define THREADS 32

// Time step size
#define DT 0.075

// Gravitational constant
#define G 100

// Relevent radii
#define CANNONBALL_RADIUS 2
#define SPACESHIP_RADIUS 4

// Relevant masses
#define CANNONBALL_MASS 4
#define SPACESHIP_MASS 16

#define CANNONBALL_EXIT_POS 10
#define CANNONBALL_EXIT_VEL 10

// Directions
#define NONE 0
#define UP 1
#define DOWN 2
#define RIGHT 3
#define LEFT 4

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


/***************************** GLOBAL VARIABLES ****************************/
// These variables should never be modified beyond their initialized values.
star_t* stars;
int num_stars;


/**************************FUNCTION IMPLEMENTATIONS*************************/

// Initialize an array of star_t-s representing the stars on the playing
// field
__host__ star_t* init_stars() {
  // Initialize the global array "stars"
  stars = (star_t*) malloc(sizeof(star_t) * 2); 

  // First star
  stars[0].mass = 400;
  stars[0].radius = 20;
  stars[0].x_position = SCREEN_WIDTH/3;
  stars[0].y_position = SCREEN_WIDTH/2;
  // Second star
  stars[1].mass = 400;
  stars[1].radius = 20;
  stars[1].x_position = 2*(SCREEN_WIDTH/3);
  stars[1].y_position = SCREEN_WIDTH/2;

  num_stars = 2; // Inititializing the global int "num_stars"

  return stars;
}

// Free the array of stars
__host__ void free_stars() {
  free(stars);
}

// Places the user's spaceship on oneside of the field, depending on whether
// the user is the first or second client to connect to the server
__host__ spaceship_t * init_spaceship(int clientID) {
  spaceship_t* spaceship = (spaceship_t*) malloc(sizeof(spaceship_t));

  spaceship->clientID = clientID;

  switch(clientID) {
    case 0 :
      spaceship->x_position = SCREEN_WIDTH/5;
      spaceship->y_position = SCREEN_HEIGHT/5;
      break;
    case 1 :
      spaceship->x_position = 4*(SCREEN_WIDTH/5);
      spaceship->y_position = 4*(SCREEN_HEIGHT/5);
      break;
  }
  return spaceship;
}

__host__ void free_spaceship(spaceship_t* spaceship) {
  free(spaceship);
}

__host__ cannonball_t* init_cannonballs() {
  cannonball_t* cannonballs = (cannonball_t*) malloc(sizeof(cannonball_t));
  return cannonballs;
}

__host__ void free_cannonballs(cannonball_t* cannonballs) {
  free(cannonballs);
}

__host__ bool is_cannonball_in_bounds(spaceship_t* spaceship,
                                      int direction_shot) {
  bool result; 
  float cannonball_x_pos;
  float cannonball_y_pos;

  switch(direction_shot) {
    case UP :
      cannonball_x_pos = spaceship->x_position;
      cannonball_y_pos = spaceship->y_position - CANNONBALL_EXIT_POS;
      break;
    case DOWN :
      cannonball_x_pos = spaceship->x_position;
      cannonball_y_pos = spaceship->y_position + CANNONBALL_EXIT_POS;
      break;
    case RIGHT :
      cannonball_x_pos = spaceship->x_position + CANNONBALL_EXIT_POS;
      cannonball_y_pos = spaceship->y_position;
      break;
    case LEFT :
      cannonball_x_pos = spaceship->x_position - CANNONBALL_EXIT_POS;
      cannonball_y_pos = spaceship->y_position;
      break;
  }

  // Is this cannonball within the bounds of the screen?
  if (cannonball_x_pos > 0 &&
      cannonball_x_pos <= SCREEN_WIDTH &&
      cannonball_y_pos > 0 &&
      cannonball_y_pos <= SCREEN_HEIGHT) {
    result = true;
  }
  else {
    result = false;
  }

  return result;
}

// Add a cannonball to the field (Note: the caller must update the number of
// cannonballs!)
__host__ cannonball_t* add_cannonball(spaceship_t* spaceship,
                                      int direction_shot,
                                      cannonball_t* cannonballs,
                                      int num_cannonballs) {
  float cannonball_x_pos;
  float cannonball_y_pos;
  float cannonball_x_vel;
  float cannonball_y_vel;


  switch(direction_shot) {
    case UP :
      cannonball_x_pos = spaceship->x_position;
      cannonball_y_pos = spaceship->y_position - CANNONBALL_EXIT_POS;
      cannonball_x_vel = spaceship->x_velocity;
      cannonball_y_vel = spaceship->y_velocity - CANNONBALL_EXIT_VEL;
      break;
    case DOWN :
      cannonball_x_pos = spaceship->x_position;
      cannonball_y_pos = spaceship->y_position + CANNONBALL_EXIT_POS;
      cannonball_x_vel = spaceship->x_velocity;
      cannonball_y_vel = spaceship->y_velocity + CANNONBALL_EXIT_VEL;
      break;
    case RIGHT :
      cannonball_x_pos = spaceship->x_position + CANNONBALL_EXIT_POS;
      cannonball_y_pos = spaceship->y_position;
      cannonball_x_vel = spaceship->x_velocity + CANNONBALL_EXIT_VEL;
      cannonball_y_vel = spaceship->y_velocity;
      break;
    case LEFT :
      cannonball_x_pos = spaceship->x_position - CANNONBALL_EXIT_POS;
      cannonball_y_pos = spaceship->y_position;
      cannonball_x_vel = spaceship->x_velocity - CANNONBALL_EXIT_VEL;
      cannonball_y_vel = spaceship->y_velocity;
      break;
  }


  // Reallocate memory to make space for the new cannonball
  cannonballs = (cannonball_t*)
    realloc(cannonballs, num_cannonballs * sizeof(cannonball_t));
  
  cannonballs[num_cannonballs].x_position = cannonball_x_pos;
  cannonballs[num_cannonballs].y_position = cannonball_y_pos;
  cannonballs[num_cannonballs].x_velocity = cannonball_x_vel;
  cannonballs[num_cannonballs].y_velocity = cannonball_y_vel;

  return cannonballs;
}


// Update position and velocity of a spaceship
__host__ spaceship_t * update_spaceship(spaceship_t* spaceship,
                                        int direction_boost) {
  spaceship->x_position += spaceship->x_velocity * DT;
  spaceship->y_position += spaceship->y_velocity * DT;

  // Loop over all stars to compute forces
  for(int j = 0; j < num_stars ; j++) {

    // Compute the distance between the cannonball and each star in each
    // dimension
    float x_diff = spaceship->x_position - stars[j].x_position;
    float y_diff = spaceship->y_position - stars[j].y_position;

    // Compute the magnitude of the distance vector
    float dist = sqrt(x_diff * x_diff + y_diff * y_diff);

    // Normalize the distance vector components
    x_diff /= dist;
    y_diff /= dist;

    // Keep a minimum distance, otherwise we get
    // Is this necessary? Could be used for collisions
    float combined_radius = SPACESHIP_RADIUS + stars[j].radius;
    if(dist < combined_radius) {
      dist = combined_radius;
    }

    // Compute the x and y accelerations
    float x_boost;
    float y_boost;
    switch(direction_boost) {
      case NONE :
        x_boost = 0;
        y_boost = 0;
        break;
      case UP :
        x_boost = 0;
        y_boost = -10;
        break;
      case DOWN :
        x_boost = 0;
        y_boost = 10;
        break;
      case RIGHT :
        x_boost = 10;
        y_boost = 0;
        break;
      case LEFT :
        x_boost = -10;
        y_boost = 0;
        break;
    }
    
    float x_acceleration = -x_diff * G * CANNONBALL_MASS / (dist * dist) +
      x_boost;
    float y_acceleration = -y_diff * G * CANNONBALL_MASS / (dist * dist) +
      y_boost;

    // Update the star velocity
    spaceship->x_velocity += x_acceleration * DT;
    spaceship->y_velocity += y_acceleration * DT;

    // Handle edge collisiosn
    if(spaceship->x_position < 0 && spaceship->x_velocity < 0)
      spaceship->x_velocity *= -0.5;
    if(spaceship->x_position >= SCREEN_WIDTH && spaceship->x_velocity > 0)
      spaceship->x_velocity *= -0.5;
    if(spaceship->y_position < 0 && spaceship->y_velocity < 0)
      spaceship->y_velocity *= -0.5;
    if(spaceship->y_position >= SCREEN_HEIGHT && spaceship->y_velocity > 0)
      spaceship->y_velocity *= -0.5;
  }
  return spaceship;
}

// Has the GPU update cannonballs and transfers them to the CPU.
__host__ void  update_cannonballs(cannonball_t* cpu_cannonballs,
                                  int num_cannonballs) {
  cannonball_t* gpu_cannonballs = NULL;

  // Realloc from cpu to gpu
  gpuErrchk(hipMalloc(&gpu_cannonballs,
                       sizeof(cannonball_t) * (num_cannonballs))); 
  gpuErrchk(hipMemcpy(gpu_cannonballs, cpu_cannonballs,
                       sizeof(cannonball_t) * (num_cannonballs),
                       hipMemcpyHostToDevice));

  int blocks = (num_cannonballs + THREADS - 1) / THREADS;

  // Calculate positions and velocities of all cannonballs in the gpu
  update_cannonballs_gpu<<<blocks, THREADS>>>(gpu_cannonballs,
                                              num_cannonballs, stars,
                                              num_stars);
  gpuErrchk(hipDeviceSynchronize());

  // Copy udated cannonballs back to CPU
  gpuErrchk(hipMemcpy(cpu_cannonballs, gpu_cannonballs,
                       sizeof(cannonball_t) * num_cannonballs,
                       hipMemcpyDeviceToHost));

  free(gpu_cannonballs);
}

// Updates cannonballs' position and velocity concurrently using the GPU
__global__ void update_cannonballs_gpu(cannonball_t* cannonballs,
                                       int num_cannonballs,
                                       star_t* stars,
                                       int num_stars) {
  int i = (blockIdx.x * THREADS) + threadIdx.x;
  if (i < num_cannonballs) {
    cannonballs[i].x_position += cannonballs[i].x_velocity * DT;
    cannonballs[i].y_position += cannonballs[i].y_velocity * DT;

    // Loop over all stars to compute forces
    for(int j = 0; j < num_stars ; j++) {
      // Don't compute the force of a star on itself
      // vvv cannonballs don't compute on themselves
      // if(i == j) continue;

      // Compute the distance between the cannonball and each star in each
      // dimension
      float x_diff = cannonballs[i].x_position - stars[j].x_position;
      float y_diff = cannonballs[i].y_position - stars[j].y_position;

      // Compute the magnitude of the distance vector
      float dist = sqrt(x_diff * x_diff + y_diff * y_diff);

      // Normalize the distance vector components
      x_diff /= dist;
      y_diff /= dist;

      // Keep a minimum distance, otherwise we get
      // Is this necessary? Could be used for collisions
      float combined_radius = CANNONBALL_RADIUS + stars[j].radius;
      if(dist < combined_radius) {
        dist = combined_radius;
      }

      // Compute the x and y accelerations
      float x_acceleration = -x_diff * G * CANNONBALL_MASS / (dist * dist);
      float y_acceleration = -y_diff * G * CANNONBALL_MASS / (dist * dist);

      // Update the star velocity
      cannonballs[i].x_velocity += x_acceleration * DT;
      cannonballs[i].y_velocity += y_acceleration * DT;

      // Handle edge collisiosn
      if(cannonballs[i].x_position < 0 && cannonballs[i].x_velocity < 0)
        cannonballs[i].x_velocity *= -0.5;
      if(cannonballs[i].x_position >= SCREEN_WIDTH
         && cannonballs[i].x_velocity > 0)
        cannonballs[i].x_velocity *= -0.5;
      if(cannonballs[i].y_position < 0 && cannonballs[i].y_velocity < 0)
        cannonballs[i].y_velocity *= -0.5;
      if(cannonballs[i].y_position >= SCREEN_HEIGHT
         && cannonballs[i].y_velocity > 0)
        cannonballs[i].y_velocity *= -0.5;
    }
  }
}


__host__ bool spaceship_collision(spaceship_t* spaceship,
                                  cannonball_t* cannonballs,
                                  int num_cannonballs) {
  float ship_x = spaceship->x_position;
  float ship_y = spaceship->y_position;

  // Check for collisions with all stars
  for (int i = 0; i < num_stars; i++) {
    if (check_collision(ship_x, ship_y, SPACESHIP_RADIUS,
                        stars[i].x_position, stars[i].y_position,
                        stars[i].radius)) {
      return true;
    }
  }
  // Check for collisions with all cannonballs
  for (int i = 0; i < num_cannonballs; i++) {
    if (check_collision(ship_x, ship_y, SPACESHIP_RADIUS,
                        cannonballs[i].x_position,
                        cannonballs[i].y_position, CANNONBALL_RADIUS)) {
      return true;
    }
  }
  // If it hasn't found a collision thusfar, then there is none
  return false;
}
   

// Is there a collision here?
__host__ bool check_collision(float obj1_x,float obj1_y,float obj1_radius,
                              float obj2_x,float obj2_y,float obj2_radius) {
  // Compute the distance between each obj in each dimension
  float x_diff = obj1_x - obj2_x;
  float y_diff = obj1_y - obj2_y;

  // Compute the magnitude of the distance vector
  float dist = sqrt(x_diff * x_diff + y_diff * y_diff);

  // If the distance between the objects is <= their combined radius, then
  // there is a collision
  if (dist <= obj1_radius + obj2_radius) {
    return true;
  } else {
    return false;
  }
}
